#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <stddef.h>
#include <sys/types.h>
#include <math.h>
#include <time.h>
#include "rngpu.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <hip/hip_runtime_api.h>
#include <omp.h>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <fstream>

#define BLOCKSPERGRID 1024
#define TILE_WIDTH 32

#ifndef THREADSPERBLOCK
#define THREADSPERBLOCK 1024
#endif
#ifndef DIM_PARAM
#define DIM_PARAM 20
#endif
#ifndef CPUITERATIONS
#define CPUITERATIONS 10
#endif
#ifndef INITIALIZATIONMODE
#define INITIALIZATIONMODE 2
#endif

// Makros taken from Christian Hundt
// https://github.com/gravitino/cudahelpers
#define CUERR { \
        hipError_t cudaerr; \
        if ((cudaerr = hipGetLastError()) != hipSuccess){ \
            printf("CUDA ERROR: \"%s\" at LINE %d.\n", hipGetErrorString(cudaerr), __LINE__); \
        } \
}

#define TIMERSTART(label)                                                    \
        hipEvent_t start##label, stop##label;                               \
        float time##label;                                                   \
        hipEventCreate(&start##label);                                      \
        hipEventCreate(&stop##label);                                       \
        hipEventRecord(start##label, 0);

#define TIMERSTOP(label)                                                     \
        hipEventRecord(stop##label, 0);                                     \
        hipEventSynchronize(stop##label);                                   \
        hipEventElapsedTime(&time##label, start##label, stop##label);       \
        printf("#%f ms (%s)\n", time##label, #label);

using namespace std;

//texture<uint, hipTextureType2D, hipReadModeElementType> texRef;

void readInputFileData(uint32_t**, uint32_t**, int*, int*, double*, string);

void readInputFileTSV(uint32_t**, uint32_t**, int*, int*, double*, string);

//void readInputFileMovieLens(uint32_t**, uint32_t**, int*, int *, double*, string);

bool endsWith(const string&, const string&);

// void initializeTextureMemory(uint32_t**, int, int);

void initializeFactors(uint32_t**, uint32_t**, uint32_t**, uint32_t**, int, int, float, fast_kiss_state32_t*);

void computeStartError(uint32_t*, uint32_t*, uint32_t*, int, int, int**, int*);

void checkDistance(uint32_t*, uint32_t*, uint32_t*, int, int);

void aftertestGPU(uint32_t*, uint32_t*, uint32_t*, int, int);

void writeToFiles(uint32_t*, uint32_t*, int, int);

void CPUcomputation(uint32_t*, uint32_t*, uint32_t*, int, int, int, uint32_t, int, float, int);

void CPUvectorMatrixMultCompareRow(uint32_t*, uint32_t*, uint32_t*, int, int, int, int*, fast_kiss_state32_t*, int);

void CPUvectorMatrixMultCompareCol(uint32_t*, uint32_t*, uint32_t*, int, int, int, int*, fast_kiss_state32_t*, int);

void aftertestCPU(uint32_t*, uint32_t*, uint32_t*, uint32_t*, uint32_t*, int, int);

__global__ void vectorMatrixMultCompareRow(uint32_t*, uint32_t*, uint32_t*, int, int, int, int*, uint32_t, float);

__global__ void vectorMatrixMultCompareCol(uint32_t*, uint32_t*, uint32_t*, int, int, int, int*, uint32_t, float);

__global__ void computeFullError(uint32_t*, uint32_t*, uint32_t*, int, int, int*);

__global__ void matrixMultiply(uint32_t*, uint32_t*, uint32_t*, int, int);

__global__ void matrixMultiplyInt(int*, int*, uint32_t*, int, int, int);

__inline__ __device__ int warpReduceSum(int);

// Main
//
int main(int argc, char **argv) {
    hipProfilerStart();
    /* ./a.out  [data]
                [updateStep]
                [lines at once]
                [threshold]
                [gpu iterations]
                [startingTemperature]
                [iterations till temperature is reduced]
    */
    if(argc == 1) return 0;
    std::string filename = argv[1];
    int updateStep = argc > 2 ? atoi(argv[2]) : 1000;
    int linesAtOnce = argc > 3 ? atoi(argv[3]) : 4000;
    float threshold = argc > 4 ? atof(argv[4]) : 0;
    int gpuiterations = argc > 5 ? atoi(argv[5]) : 10000;
    int maxIterationsNoImp = argc > 6 ? atoi(argv[6]) : 10000;
    float temperature = argc > 7 ?  atof(argv[7]) : 0;
    int iterationsTillReduced = argc > 8 ? (atoi(argv[8]) > 0? atoi(argv[8]) : INT_MAX) : INT_MAX;
    float tempFactor = argc > 9 ? atof(argv[9]) : 0.99;
    int seed = argc > 10 ? atoi(argv[10]) : 41;
    fast_kiss_state32_t state = get_initial_fast_kiss_state32(seed);
    
    // Discard first 100000 entries of PRNG
    for (int i = 0; i < 100000; i++)
        fast_kiss32(&state);
    
    // Read file and save matrix in C0 and d_C0
    uint32_t *C0, *d_C0;
    int width, height;
    double density;
    
    // Dense TSV
    std::string ending = "tsv";
    if (endsWith(filename, ending)) {
        readInputFileTSV(&C0, &d_C0, &width, &height, &density, filename);
    } 
    
    // COO coordinates
    ending = "dat";
    if (endsWith(filename, ending)) {
        readInputFileData(&C0, &d_C0, &width, &height, &density, filename);
    }
        
    // Initialize Texture Memory with C0
    // initializeTextureMemory(&C0, width, height);

    // Initialize Ab, Bb, d_Ab, d_Bb, all bitwise used matrices
    uint32_t *Ab, *Bb;
    uint32_t *d_Ab, *d_Bb;
    initializeFactors(&Ab, &Bb, &d_Ab, &d_Bb, width, height, density, &state);
    // A and B now initialized on device and host

    // Calculate original error, save it two times each one for GPU, one for CPU
    int error_C0_C_start = 0;
    int error_C0_C       = 0;
    int *d_error_C0_C_start, *d_error_C0_C;
    hipMalloc((void **) &d_error_C0_C, sizeof(int));                                                           CUERR
    hipMalloc((void **) &d_error_C0_C_start, sizeof(int));                                                     CUERR
    hipMemcpy(d_error_C0_C_start, &error_C0_C_start, sizeof(int), hipMemcpyHostToDevice);                     CUERR
    
    computeStartError(d_Ab, d_Bb, d_C0, width, height, &d_error_C0_C_start, &error_C0_C_start);
    
    hipMemcpy(d_error_C0_C, d_error_C0_C_start, sizeof(int), hipMemcpyDeviceToDevice);
    error_C0_C = error_C0_C_start;
    // Now the starting errors is in stored in 4 values
    // error_C0_C_start, error_C0_C on CPU and GPU


    // MAIN PART
    // on GPU
    int iterations = 0;
    int toBeChanged;
    int iterationsNoImp = 0;
    int error_C0_C_before = 0;
    threshold *= (width*height);
    #ifdef PERF
    vector<double> errorVector; // vector for error measurement
    vector<int> impVector;
    updateStep = 1;
    #endif
    printf("\n- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");
    printf("- Starting %i GPU iterations, pulling error every %i steps -\n", gpuiterations, updateStep);
    printf("- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");
    TIMERSTART(GPUKERNELLOOP)
    while (threshold < error_C0_C && iterations < gpuiterations && iterationsNoImp < maxIterationsNoImp) {
        iterations++;
        if (iterations % iterationsTillReduced == 0)
            temperature *= tempFactor;
        
        // Pull error from GPU to show it
        if (iterations % updateStep == 0) {
            #ifndef PERF
            printf("Current error: %f\n", error_C0_C / (double) (width * height));
            #endif
            if (error_C0_C_before - error_C0_C == 0) {
                iterationsNoImp += updateStep;
            } else {
                #ifdef PERF
                impVector.push_back(iterationsNoImp);
                #endif
                iterationsNoImp = 0;
            }
            error_C0_C_before = error_C0_C;
            #ifdef PERF
            errorVector.push_back(error_C0_C / (double) (width * height));
            #endif
            //checkDistance(d_Ab, d_Bb, d_C0, height, width);
        }

        // Change col
        toBeChanged = ((unsigned int) fast_kiss32(&state)) % width;
        vectorMatrixMultCompareCol <<< min(linesAtOnce, width), THREADSPERBLOCK >>>
                                        (d_Ab, d_Bb, d_C0, width, height, toBeChanged, d_error_C0_C, 
                                        ((fast_kiss32(&state) + iterations) % UINT32_MAX), temperature);        CUERR
        hipDeviceSynchronize();                                                                                CUERR

        // Change row
        toBeChanged = ((unsigned int) fast_kiss32(&state)) % height;
        vectorMatrixMultCompareRow <<< min(linesAtOnce, height), THREADSPERBLOCK >>>
                                        (d_Ab, d_Bb, d_C0, width, height, toBeChanged, d_error_C0_C, 
                                        ((fast_kiss32(&state) + iterations) % UINT32_MAX), temperature);        CUERR
        hipDeviceSynchronize();                                                                                CUERR
        
        hipMemcpy(&error_C0_C, d_error_C0_C, sizeof(int), hipMemcpyDeviceToHost);                             CUERR
    }
    // Pull final error from GPU
    hipMemcpy(&error_C0_C, d_error_C0_C, sizeof(int), hipMemcpyDeviceToHost);                                 CUERR
    printf("- - - - - - - - -\n");
    printf("Final Error on GPU: %f, %i wrong entries\n", error_C0_C / (double) (height * width), error_C0_C);
    TIMERSTOP(GPUKERNELLOOP)
    
    // Aftertest GPU
    #ifdef TEXT
    //aftertestGPU(d_Ab, d_Bb, d_C0, width, height);
    #endif
    
    writeToFiles(d_Ab, d_Bb, width, height);

    
    #ifdef PERF
    string writeFile = string("perf.csv");
    ofstream myfile(writeFile);
    if (myfile.is_open()) {
        myfile << "x,y\n";
        for (int i = 0; i < errorVector.size(); i++) {
            myfile << (i * timeGPUKERNELLOOP / (double) iterations) / (double) 1000 << "," << errorVector[i] << "\n";            
        }
    }
    writeFile = string("update.csv");
    ofstream myfile1(writeFile);
    if (myfile1.is_open()) {
        myfile1 << "x,y\n";
        for (int i = 0; i < impVector.size(); i++) {
            myfile1 << i << "," << impVector[i] << "\n";        
        }
    }
    #endif

    #ifdef CPU
    // CPU COMPUTATION
    //CPUcomputation(Ab, Bb, C0, width, height, error_C0_C_start, 42, updateStep, threshold, linesAtOnce);
    // Aftertest CPU
    //aftertestCPU(Ab, Bb, d_Ab, d_Bb, C0, width, height);
    #endif

    printf("- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");


    // Cleaning up
    hipProfilerStop();
    hipDeviceReset();
    free(Ab);
    free(Bb);
    free(C0);
    hipFree(d_Ab);
    hipFree(d_Bb);
    hipFree(d_C0);
    hipFree(d_error_C0_C);
    hipFree(d_error_C0_C_start);
    return 0;
}

__inline__ __device__
int warpReduceSum(int val) {
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
        val += __shfl_down(val, offset);
    return val;
}

// [A] row Change kernel
__global__ void
vectorMatrixMultCompareRow( uint32_t *A, uint32_t *B, uint32_t *C, 
                            int width, int height, 
                            int startrow, int *global_error,
                            uint32_t seed, float temperature) {

    int rowToBeChanged = (blockIdx.x + startrow) % height;
    int cTruthEntry;
    int cEntryOld;
    int cEntryNew;
    int error_thread;
    //int intId;
    //int intLane;
    uint32_t randomNumber;
    //uint32_t currentColThread;
    //uint32_t currentRow;
    //uint32_t currentRow_changed;
    float metro;
    __shared__ fast_kiss_state32_t state;
    __shared__ int reductionArray[32];
    //__shared__ uint32_t shared_currentRow_changed;
    __shared__ uint32_t shared_currentRow[DIM_PARAM];
    __shared__ uint32_t shared_currentRow_changed[DIM_PARAM];
    
    //currentRow = A[rowToBeChanged];
    if (threadIdx.x == 0) {
        //shared_currentRow_changed = currentRow; // load row to be changed in shared memory
        
        state = get_initial_fast_kiss_state32((seed + blockIdx.x) % UINT32_MAX);
        randomNumber = fast_kiss32(&state);
        #pragma unroll
        for (int i = 0; i < DIM_PARAM; i++){
            shared_currentRow[i] = A[(rowToBeChanged*DIM_PARAM) + i];
            shared_currentRow_changed[i] = shared_currentRow[i];
            shared_currentRow_changed[i] ^= (randomNumber >> i) & 11 ? 0 : 1;
            // shared_currentRow_changed ^= (randomNumber >> i) & 11 ? (0 << i) : (1 << i);
        }
    }
    __syncthreads();
    
    // currentRow_changed = shared_currentRow_changed;
    error_thread = 0;
    for (int i = 0; i <= ((width - 1) / blockDim.x); i++) {
        if ((i * blockDim.x + threadIdx.x) < width) {
            //currentColThread = B[i * blockDim.x + threadIdx.x];
            //intId = (((i * blockDim.x + threadIdx.x) * height) + rowToBeChanged) / 32;
            //intLane = (((i * blockDim.x + threadIdx.x) * height) + rowToBeChanged) % 32;
            //cTruthEntry = (C[intId] >> 32 - intLane - 1) & 1; 
            
            cTruthEntry = C[rowToBeChanged*width + (i*blockDim.x) + threadIdx.x];

            cEntryOld = cEntryNew = 0;
            for (int j = 0; j < DIM_PARAM; j++){
                if (cEntryOld < 0.5)
                    cEntryOld += shared_currentRow[j] * B[(j*width) + (i*blockDim.x + threadIdx.x)];                
                if (cEntryNew < 0.5)
                    cEntryNew += shared_currentRow_changed[j] * B[(j*width) + (i*blockDim.x + threadIdx.x)];
            }
            //cEntryOld = (currentRow         & currentColThread) > 0 ? 1 : 0;
            //cEntryNew = (currentRow_changed & currentColThread) > 0 ? 1 : 0;
            error_thread += ((cEntryNew - cTruthEntry) * (cEntryNew - cTruthEntry)) -
                            ((cEntryOld - cTruthEntry) * (cEntryOld - cTruthEntry));
        }
    }
    __syncthreads();

    // Reduction across block
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;
    error_thread = warpReduceSum(error_thread);
    if (lane == 0) reductionArray[wid] = error_thread;
    __syncthreads();
    error_thread = (threadIdx.x < blockDim.x / warpSize) ? reductionArray[lane] : 0;
    if (wid == 0) error_thread = warpReduceSum(error_thread);
    // Thread with threadIdx.x==0 now has error in error_thread

    // Thread 0 checks if new low has been found and applies if necessary
    if (threadIdx.x == 0) {
        if (error_thread < 0) {
             //A[rowToBeChanged] = shared_currentRow_changed;
            for(int i=0; i<DIM_PARAM; i++)
                A[rowToBeChanged*DIM_PARAM+i] = shared_currentRow_changed[i];
            atomicAdd(global_error, error_thread);
        } else { // Metropolis–Hastings algorithm
            randomNumber = fast_kiss32(&state) / (double) UINT32_MAX;
            metro = temperature > 0.0f ? fminf(1, expf(-error_thread / temperature)) : 0 ;
            if (randomNumber < metro) {
                // A[rowToBeChanged] = shared_currentRow_changed;
                for(int i=0; i<DIM_PARAM; i++)
                    A[rowToBeChanged*DIM_PARAM+i] = shared_currentRow_changed[i];
                atomicAdd(global_error, error_thread);
            }
        }
    }
}

// [B] col change kernel
//
__global__ void
vectorMatrixMultCompareCol(uint32_t *A, uint32_t *B, uint32_t *C, 
                            int width, int height, 
                            int startcol, int *global_error,
                            uint32_t seed, float temperature) {

    int colToBeChanged = (blockIdx.x + startcol) % width;
    int cTruthEntry;
    int cEntryOld;
    int cEntryNew;
    int error_thread;
    //int intId;
    //int intLane;
    uint32_t randomNumber;
    //uint32_t currentRowThread;
    //uint32_t currentCol;
    //uint32_t currentCol_changed;
    float metro;
    __shared__ fast_kiss_state32_t state;
    __shared__ int shared[32];
    //__shared__ uint32_t shared_currentCol_changed;
    __shared__ uint32_t shared_currentCol[DIM_PARAM];
    __shared__ uint32_t shared_currentCol_changed[DIM_PARAM];

    //currentCol = B[colToBeChanged];
    if (threadIdx.x == 0) {
        //shared_currentCol_changed = currentCol; // load row to be changed in shared memory

        state = get_initial_fast_kiss_state32((seed + blockIdx.x) % UINT32_MAX);
        randomNumber = fast_kiss32(&state);
        #pragma unroll
        for (int i = 0; i < DIM_PARAM; i++){
            shared_currentCol[i] = B[i*width + colToBeChanged];
            shared_currentCol_changed[i] = shared_currentCol[i];
            shared_currentCol_changed[i] ^= (randomNumber >> i) & 11 ? 0 : 1;
            //shared_currentCol_changed ^= (randomNumber >> i) & 11 ? (0 << i) : (1 << i);
        }
    }
    __syncthreads();
    
    //currentCol_changed = shared_currentCol_changed;
    error_thread = 0;
    for (int i = 0; i <= ((height - 1) / blockDim.x); i++) {
        if ((i * blockDim.x + threadIdx.x) < height) {
            //currentRowThread = A[i * blockDim.x + threadIdx.x];
            //intId = ((colToBeChanged * height) + (i * blockDim.x + threadIdx.x)) / 32;
            //intLane = ((colToBeChanged * height) + (i * blockDim.x + threadIdx.x)) % 32;
            //cTruthEntry = (C[intId] >> 32 - intLane - 1) & 1; 
            
			cTruthEntry = C[(i*blockDim.x + threadIdx.x)*width + colToBeChanged];
            
            cEntryOld = cEntryNew = 0;
            for (int j = 0; j < DIM_PARAM; j++) {
                if (cEntryOld < 0.5)
                    cEntryOld += shared_currentCol[j] * A[(blockDim.x*i + threadIdx.x) * DIM_PARAM + j];
                if (cEntryNew < 0.5)
                    cEntryNew += shared_currentCol_changed[j] * A[(blockDim.x*i + threadIdx.x) * DIM_PARAM + j];
            }
            
            //cEntryOld = (currentCol         & currentRowThread) > 0 ? 1 : 0;        
            //cEntryNew = (currentCol_changed & currentRowThread) > 0 ? 1 : 0;
            error_thread += ((cEntryNew - cTruthEntry) * (cEntryNew - cTruthEntry)) -
                            ((cEntryOld - cTruthEntry) * (cEntryOld - cTruthEntry));
        }
    }
    __syncthreads();

    // Reduction across block
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;
    error_thread = warpReduceSum(error_thread);
    if (lane == 0) shared[wid] = error_thread;
    __syncthreads();
    error_thread = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;
    if (wid == 0) error_thread = warpReduceSum(error_thread);
    // Thread with threadIdx.x==0 now has error in error_thread

    // Thread 0 checks if new low has been found and applies if necessary
    if (threadIdx.x == 0) {
        if (error_thread < 0) {
            //B[colToBeChanged] = shared_currentCol_changed;
            for(int i=0; i<DIM_PARAM; i++)
                B[i*width + colToBeChanged] = shared_currentCol_changed[i];
            atomicAdd(global_error, error_thread);
        }  else { // Metropolis–Hastings algorithm
            randomNumber = fast_kiss32(&state) / (double) UINT32_MAX;
            metro = temperature > 0.0f ? fminf(1, expf(-error_thread / temperature)) : 0 ;
            if (randomNumber < metro) {
				for(int i=0; i<DIM_PARAM; i++)
					B[i*width + colToBeChanged] = shared_currentCol_changed[i];
                //B[colToBeChanged] = shared_currentCol_changed;
                atomicAdd(global_error, error_thread);
            }
        }
    }
    __syncthreads();
}

// Start error kernel
__global__ void computeFullError(   uint32_t *A, uint32_t *B, uint32_t *C, 
                                    int width, int height, int *distance_test) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int lineSum;
    int truthEntry;
    int intId;
    int intLane;
    int error_thread;
    //__shared__ volatile int shared_distance[THREADSPERBLOCK];
    //shared_distance[threadIdx.x] = 0;
    __shared__ int reductionArray[32];
    
    
    if (tid < width) {
        error_thread = 0;
        for (int j = 0; j < height; j++) {
            lineSum = 0;
            for (int i = 0; i < DIM_PARAM; i++) {
				if(lineSum < 0.5)
					lineSum += (A[j*DIM_PARAM + i]) * (B[(i*width) + tid]);
            }
            //lineSum = (A[j] & B[tid]) > 0 ? 1 : 0;
            //intId = (tid * height + j) / 32;
            //intLane = (tid * height + j) % 32;
            
            //truthEntry = (C[intId] >> 32 - intLane - 1) & 1; 
            //if(A[j*DIM_PARAM + i] * B[(i*width) + tid] < 0)
            //    printf("BALBAL \t");
            truthEntry = C[j*width + tid];
            
            error_thread += ((lineSum - truthEntry) * (lineSum - truthEntry));
        }
        __syncthreads();
        //printf("Own error: %d\n", error_thread);
        
        // Reduction across block
        int lane = threadIdx.x % warpSize;
        int wid = threadIdx.x / warpSize;
        error_thread = warpReduceSum(error_thread);
        if (lane == 0) reductionArray[wid] = error_thread;
        __syncthreads();
        error_thread = (threadIdx.x < blockDim.x / warpSize) ? reductionArray[lane] : 0;
        if (wid == 0) error_thread = warpReduceSum(error_thread);
        // Thread with threadIdx.x==0 now has error in error_thread

        if (threadIdx.x == 0)
            atomicAdd(distance_test, error_thread);
        __syncthreads();
        
    }
}

// Each thread one entry of a row
__global__ void matrixMultiply( uint32_t *A, uint32_t *B, uint32_t *C, int width, int height) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < width)
        for (int i = 0; i < height; i++)
            C[i * width + tid] = (A[i] & B[tid]) > 0 ? 1 : 0;
}

__global__ void matrixMultiplyInt(  int * A0, int * B0, uint32_t * C0, 
                                    int m, int k, int n) {
    __shared__ int ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ int ds_B[TILE_WIDTH][TILE_WIDTH];
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int sum = 0;

    for(int t = 0; t < (n - 1) / TILE_WIDTH + 1; t++) {
        if(row < m && t * TILE_WIDTH + tx < n)
            ds_A[ty][tx] = roundf(A0[row * n + t * TILE_WIDTH + tx]);
        else
            ds_A[ty][tx] = 0.0;
        if(t * TILE_WIDTH + ty < n && col < k)
            ds_B[ty][tx] = roundf(B0[(t * TILE_WIDTH + ty) * k + col]);
        else
            ds_B[ty][tx] = 0.0;
        __syncthreads();
        for(int i = 0; i < TILE_WIDTH; i++){
            sum += ds_A[ty][i] * ds_B[i][tx];
        }
        __syncthreads();
    }
    if(row < m && col < k)
        C0[col + row * k] = min(1, sum);

}

void readInputFileData( uint32_t **C0, uint32_t **d_C0, 
                    int *width, int *height, 
                    double *density, string filename) {
    int x, y;
    int nonzeroelements = 0;
    //int intID;
    //int intLane;
    ifstream infile;
    string linestring;
    string field;

    // First line: #height,#width,#non-zero-elements
    infile.open(filename);
    getline(infile, linestring);
    stringstream sep(linestring);
    getline(sep, field, ',');
    (*height) = stoi(field, nullptr);
    getline(sep, field, '\n');
    (*width) = stoi(field, nullptr);
    
    // Malloc for C0 and d_C0
    int sizeC = ((*width) * (*height));
    //int sizeC = (int) ceil((*width) * (*height) / (double) 32.0);
    (*C0) = (uint32_t *) malloc(sizeof(uint32_t) * sizeC);
    hipMalloc((void **) d_C0, sizeof(uint32_t) * sizeC);                                       CUERR
    
    // Set all entries 0
    for (int i = 0; i < sizeC; i++)
        (*C0)[i] = 0;

    // Read rest of file
    while (getline(infile, linestring)) {
        stringstream sep1(linestring);
        string fieldtemp;
        getline(sep1, fieldtemp, ',');
        y = stoi(fieldtemp, nullptr);
        getline(sep1, fieldtemp, ',');
        x = stoi(fieldtemp, nullptr);
        //intID = (x * (*height) + y) / 32;
        //intLane = (x * (*height) + y) % 32;
        //(*C0)[intID] |= 1 << 32 - intLane - 1;
        (*C0)[y * (*width) + x] = 1;
        nonzeroelements++;
    }
    
    (*density) = (double) nonzeroelements / ((*width) * (*height));

    hipMemcpy((*d_C0), (*C0), sizeof(uint32_t) * sizeC, hipMemcpyHostToDevice);               CUERR
       
    printf("- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");
    printf("READING OF .DATA FILE COMPLETE\n");
    printf("Read height: %i\nRead width: %i\nNon-zero elements: %i\nDensity: %f\n",
           (*height), (*width), nonzeroelements, (*density));
    printf("- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");
}


void readInputFileTSV( uint32_t **C0, uint32_t **d_C0, 
                    int *width, int *height, 
                    double *density, string filename) {


    int intID;
    int intLane;
    int nonzeroelements;
    ifstream input(filename);
    char const row_delim = '\n';
    char const field_delim = '\t';
    vector<uint32_t> x_values;
    vector<uint32_t> y_values;
    int x_counter = 0;
    int y_counter = 0;
    
    // Read file
    for (string row; getline(input, row, row_delim); ) {
        x_counter = 0;
        istringstream ss(row);
        for (string field; getline(ss, field, field_delim); ) {
            if (stoi(field) == 1) {
                x_values.push_back(x_counter);
                y_values.push_back(y_counter);
            }
            x_counter++;
        }
        y_counter++;
    }
    *width = x_counter;
    *height = y_counter;
    nonzeroelements = x_values.size();
    (*density) = (double) nonzeroelements / ((*width) * (*height));
    
    
    // Malloc for C0 and d_C0
    int sizeC = (int) ceil((*width) * (*height));
    //int sizeC = (int) ceil((*width) * (*height) / (double) 32.0);
    (*C0) = (uint32_t *) malloc(sizeof(uint32_t) * sizeC);
    hipMalloc((void **) d_C0, sizeof(uint32_t) * sizeC);                                       CUERR
    
    // Set all entries 0
    for (int i = 0; i < sizeC; i++)
        (*C0)[i] = 0;

    // Read rest of file
    for (int i = 0; i < x_values.size(); i++) {
        intID = (x_values[i] * (*height) + y_values[i]) / 32;
        intLane = (x_values[i] * (*height) + y_values[i]) % 32;
        //(*C0)[intID] |= 1 << 32 - intLane - 1;
        (*C0)[y_values[i] * (*width) + x_values[i]] = 1;
    }
    hipMemcpy((*d_C0), (*C0), sizeof(uint32_t) * sizeC, hipMemcpyHostToDevice);               CUERR
    
    printf("- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");
    printf("READING OF DENSE .TSV FILE COMPLETE\n");
    printf("Read height: %i\nRead width: %i\nNon-zero elements: %i\nDensity: %f\n",
           (*height), (*width), nonzeroelements, (*density));
    printf("- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");
}

// https://stackoverflow.com/questions/874134/find-if-string-ends-with-another-string-in-c
bool endsWith(const string& s, const string& suffix) {
    return s.rfind(suffix) == (s.size()-suffix.size());
}

void computeStartError(uint32_t *d_Ab, uint32_t *d_Bb, uint32_t *d_Cb, 
                        int width, int height,
                        int **d_distance_C0_C_start, int *distance_C0_C_start) {
    TIMERSTART(ERRORFIRST)
    
    computeFullError <<< width / THREADSPERBLOCK + 1, THREADSPERBLOCK >>>
                        (d_Ab, d_Bb, d_Cb, width, height, (*d_distance_C0_C_start));                    CUERR
                        
    hipMemcpy(distance_C0_C_start, (*d_distance_C0_C_start), sizeof(int), hipMemcpyDeviceToHost);     CUERR

    printf("Starting error between AxB=C and C0: %f \n", 
            (*distance_C0_C_start) / ((double) width * height));
    TIMERSTOP(ERRORFIRST)
    printf("- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");
}

// Only for debugging
void checkDistance(uint32_t *d_Ab, uint32_t *d_Bb, uint32_t *d_C0, int height, int width) {
    int distance_test;
    int *d_distance_test;
    hipMalloc((void **) &d_distance_test, sizeof(int));                                                CUERR
    distance_test = 0;
    hipMemcpy(d_distance_test, &distance_test, sizeof(int), hipMemcpyHostToDevice);                   CUERR
    
    computeFullError <<< width / THREADSPERBLOCK + 1, THREADSPERBLOCK >>>  
                            (d_Ab, d_Bb, d_C0, height, width, d_distance_test);                         CUERR
                                            
    hipMemcpy(&distance_test, d_distance_test, sizeof(int), hipMemcpyDeviceToHost);                   CUERR
    printf("Real Error: %f\n", (distance_test/(double)(height*width)));
}

// Initialization of A and B
void initializeFactors( uint32_t **Ab, uint32_t **Bb, 
                        uint32_t **d_Ab, uint32_t **d_Bb, 
                        int width, int height, 
                        float density, fast_kiss_state32_t *state) {

    //(*Ab) = (uint32_t *) malloc(sizeof(uint32_t) * height);
    //(*Bb) = (uint32_t *) malloc(sizeof(uint32_t) * width);
    //hipMalloc((void **) d_Ab, sizeof(uint32_t) * height);                                              CUERR
    //hipMalloc((void **) d_Bb, sizeof(uint32_t) * width);                                               CUERR
    (*Ab) = (uint32_t *) malloc(sizeof(uint32_t) * height * DIM_PARAM);
    (*Bb) = (uint32_t *) malloc(sizeof(uint32_t) * width * DIM_PARAM);
    hipMalloc((void **) d_Ab, sizeof(uint32_t) * height * DIM_PARAM);                                  CUERR
    hipMalloc((void **) d_Bb, sizeof(uint32_t) * width * DIM_PARAM);                                   CUERR

    // Initialize A and B and copy to device
    bool threshold;
    for (int i = 0; i < height; i++) {
        //(*Ab)[i] = 0;
        #pragma unroll
        for (int j = 0; j < DIM_PARAM; j++) {
            switch(INITIALIZATIONMODE) {
                case 1: threshold = (fast_kiss32(state) / (double) UINT32_MAX) 
                                        < (sqrt(1 - pow(1 - density, 1 / (double) DIM_PARAM)));
                                        break;
                case 2: threshold = (fast_kiss32(state) / (double) UINT32_MAX) 
                                        < (density / (double) 100);
                                        break;
                case 3: threshold = (fast_kiss32(state) / (double) UINT32_MAX) 
                                        < density;
                                        break;
            }
            //(*Ab)[i] |= threshold ? 1 << (DIM_PARAM - j - 1) : 0 ;
            (*Ab)[i * DIM_PARAM + j] = threshold ? 1 : 0 ;
        }
    }
    for (int i = 0; i < width; i++) {
        //(*Bb)[i] = 0;
        #pragma unroll
        for (int j = 0; j < DIM_PARAM; j++) {
            switch(INITIALIZATIONMODE) {
                case 1: threshold = (fast_kiss32(state) / (double) UINT32_MAX) 
                                        < (sqrt(1 - pow(1 - density, 1 / (double) DIM_PARAM)));
                                        break;
                case 2: threshold = (fast_kiss32(state) / (double) UINT32_MAX) 
                                        < (density / (double) 100);
                                        break;
                case 3: threshold = (fast_kiss32(state) / (double) UINT32_MAX) 
                                        < density;
                                        break;
            }
            //(*Bb)[i] |= threshold ? 1 << (DIM_PARAM - j - 1) : 0 ;
            (*Bb)[j * width + i] = threshold ? 1 : 0 ;
        }
    }
    
    // copy to device arrays
    //hipMemcpy((*d_Ab), (*Ab), sizeof(uint32_t) * height, hipMemcpyHostToDevice);                      CUERR
    //hipMemcpy((*d_Bb), (*Bb), sizeof(uint32_t) * width, hipMemcpyHostToDevice);                       CUERR    
    hipMemcpy((*d_Ab), (*Ab), sizeof(uint32_t) * height * DIM_PARAM, hipMemcpyHostToDevice);                      CUERR
    hipMemcpy((*d_Bb), (*Bb), sizeof(uint32_t) * width * DIM_PARAM, hipMemcpyHostToDevice);                       CUERR

    printf("Initialization of A and B complete\n");
    printf("- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");
}

// Used for debugging and checking for correctness, not optimized
// NOT USED, WRONG
void aftertestGPU(  uint32_t *d_Ab, uint32_t *d_Bb, uint32_t *d_C0b, 
                    int width, int height) {
    TIMERSTART(aftertestGPU)
    uint32_t *d_C_test_GPU;
    uint32_t *C_test_GPU;
    
    int *A, *B, *C0;
    uint32_t *Ab, *Bb, *C0b;
    int *d_A, *d_B;
    float densityA, densityB;
    uint32_t counterDensity;
    A = (int*) malloc(sizeof(int) * DIM_PARAM * height);
    Ab = (uint32_t*) malloc(sizeof(uint32_t) * height);
    B = (int*) malloc(sizeof(int) * width * DIM_PARAM);
    Bb = (uint32_t*) malloc(sizeof(uint32_t) * width);
    C0 = (int*) malloc(sizeof(int) *width * height);
    C0b = (uint32_t*) malloc(sizeof(uint32_t) * ((long long) (height * width) / 32.0 + 1));
    
    hipMalloc((void**)&d_A, sizeof(int) * DIM_PARAM * height);                                             CUERR
    hipMalloc((void**)&d_B, sizeof(int) * width * DIM_PARAM);                                              CUERR
    
    hipMemcpy(Ab, d_Ab, sizeof(uint32_t) * height, hipMemcpyDeviceToHost);                              CUERR
    hipMemcpy(Bb, d_Bb, sizeof(uint32_t) * width, hipMemcpyDeviceToHost);                               CUERR
    hipMemcpy(C0b, d_C0b, sizeof(uint32_t) * ((long long)(height*width) / 32.0 + 1),
                    hipMemcpyDeviceToHost);                                                            CUERR

    counterDensity = 0;
    for(int i = 0; i < height; i++){
        for(int j = 0; j < DIM_PARAM; j++){
            A[i * DIM_PARAM + j] = (Ab[i] >> DIM_PARAM - j - 1) & 1;
            if (A[i*DIM_PARAM + j]) counterDensity++;
        }
    }
    densityA = counterDensity / (double) (height*DIM_PARAM);
    
    counterDensity = 0;
    for(int i = 0; i < width; i++) {
        for(int j = 0; j < DIM_PARAM; j++) {
            B[j * width + i] = (Bb[i] >> DIM_PARAM - j - 1) & 1;
            if (B[j * width + i]) counterDensity++;
        }
    }
    densityB = counterDensity / (double) (DIM_PARAM * width);

    int intId;
    int intLane;
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < height; j++) {
             intId = (i * height + j) / 32;
             intLane = (i * height + j) % 32;
             C0[j * width + i] = (C0b[intId] >> 32 - intLane - 1) & 1;
        }
    }
    
    hipMemcpy(d_A, A, sizeof(int)*height*DIM_PARAM, hipMemcpyHostToDevice);                           CUERR
    hipMemcpy(d_B, B, sizeof(int)*width*DIM_PARAM, hipMemcpyHostToDevice);                            CUERR   
    
    // Doing a check two times: once with A,B and once with Ab,Bb just to make sure
    // First check
    C_test_GPU = (uint32_t *) malloc(sizeof(uint32_t) * width * height);
    hipMalloc((void **) &d_C_test_GPU, sizeof(uint32_t) * height * width);                             CUERR
    
    matrixMultiply <<< width / THREADSPERBLOCK + 1, THREADSPERBLOCK >>> 
                        (d_Ab, d_Bb, d_C_test_GPU, width, height);                                      CUERR
                        
    hipMemcpy(C_test_GPU, d_C_test_GPU, sizeof(uint32_t) * height * width, 
                    hipMemcpyDeviceToHost);                                                            CUERR
    
    int error_test_GPU = 0;
    for (int i = 0; i < height * width; i++)
        error_test_GPU += (((C0[i] - C_test_GPU[i]) * (C0[i] - C_test_GPU[i])));

    // Second check
    dim3 dimGrid((width - 1) / TILE_WIDTH + 1, (height - 1) / TILE_WIDTH + 1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    
    matrixMultiplyInt <<< dimGrid, dimBlock >>> 
                            (d_A, d_B, d_C_test_GPU, height, width, DIM_PARAM);                         CUERR

    hipMemcpy(C_test_GPU, d_C_test_GPU, sizeof(int) * height * width, hipMemcpyDeviceToHost);         CUERR
    int error_test_GPU_2 = 0;
    for (int i = 0; i < height * width; i++)
        error_test_GPU_2 += (((C0[i] - C_test_GPU[i]) * (C0[i] - C_test_GPU[i])));
    
    TIMERSTOP(aftertestGPU)
    printf("Aftertest error between C0 and C on GPU (bitwise): %i\n", error_test_GPU);
    printf("Aftertest error between C0 and C on GPU (float): %i\n", error_test_GPU_2);
    printf("Density A: %f, Density B: %f\n", densityA, densityB);
    
    //writeToFiles(A, B, width, height);
}

// Write result matrix in file
void writeToFiles(uint32_t* d_A, uint32_t* d_B, int width, int height){
    uint32_t *A, *B;
    A = (uint32_t*) malloc(sizeof(uint32_t) * DIM_PARAM * height);
    B = (int*) malloc(sizeof(int) * width * DIM_PARAM);
    
    hipMemcpy(A, d_A, sizeof(uint32_t) * height * DIM_PARAM, hipMemcpyDeviceToHost);                              CUERR
    hipMemcpy(B, d_B, sizeof(uint32_t) * width * DIM_PARAM, hipMemcpyDeviceToHost);                               CUERR

    time_t rawtime;
    struct tm * timeinfo;
    char buffer[80];

    time (&rawtime);
    timeinfo = localtime(&rawtime);

    strftime(buffer, sizeof(buffer), "%X", timeinfo);
    string str = buffer;
    
    string a = string("A_") + buffer + string(".txt");
    string b = string("B_") + buffer + string(".txt");
    
    ofstream myfile(a);
    if (myfile.is_open()){
        myfile << height << "," << DIM_PARAM << "\n";
        for (int i = 0; i < height; i++){
            for (int j = 0; j < DIM_PARAM; j++){
                myfile << A[i * DIM_PARAM + j] << ((j != DIM_PARAM - 1) ? "," : "");
            }
            myfile << "\n";
        }
        myfile.close();
    }
    
    ofstream myfile2(b);
    if(myfile2.is_open()){
        myfile2 << DIM_PARAM << "," << width << "\n";
        for (int i = 0; i<DIM_PARAM; i++){
            for (int j = 0; j < width; j++){
                myfile2 << B[i * width + j] << ((j != width - 1) ? "," : "");
            }
            myfile2 << "\n";
        }
        myfile2.close();
    }   
    cout << "Writing to files \"" << a << "\" and \"" << b << "\" complete" << endl;
}

// CPU computation
void CPUcomputation(uint32_t *Ab, uint32_t *Bb, uint32_t *C0, 
                    int width, int height, 
                    int startDistance, uint32_t seed, int updateStep,
                    float threshold, int rowsAtOnce) {
                        
    int *hDistance = &startDistance;
    fast_kiss_state32_t state;
    state = get_initial_fast_kiss_state32(seed);
    int toBeChanged;
    int iterations = 0;
    TIMERSTART(CPUcomputation)
    printf("\n- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");
    printf("- - - - Starting CPU opimization, showing error every %i steps - - - - -\n", updateStep);
    printf("- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -\n");
    while (*hDistance > threshold && iterations < CPUITERATIONS) {
        if (iterations % updateStep == 0)
            printf("Current Distance: %i\n", *hDistance);
        
        // Change row
        toBeChanged = ((unsigned int) fast_kiss32(&state)) % height;
        CPUvectorMatrixMultCompareRow(Ab, Bb, C0, width, height, toBeChanged, hDistance, &state, rowsAtOnce);
        
        // Change col
        toBeChanged = ((unsigned int) fast_kiss32(&state)) % width;
        CPUvectorMatrixMultCompareCol(Ab, Bb, C0, width, height, toBeChanged, hDistance, &state, rowsAtOnce);
        iterations++;
    }
    printf("- - - - - - - - -\n");
    printf("End Distance on CPU: %i, Number of Iterations: %i,  Error remaining: %f\n", 
                *hDistance, iterations, *hDistance / (double) (height * width));
    TIMERSTOP(CPUcomputation)
}

void CPUvectorMatrixMultCompareRow( uint32_t *Ab, uint32_t *Bb, 
                                    uint32_t *C0, int width, int height, int startrow,
                                    int *hDistance, fast_kiss_state32_t *state, int rowsAtOnce) {
    int rowToBeChanged = startrow;
    int error;
    int cTruthEntry;
    int cEntryOld;
    int cEntryNew;
    uint32_t currentRow;
    uint32_t currentRow_changed;
    uint32_t randomNumber;

    // Change multiple lines from A
    for (int l = 0; l < rowsAtOnce; l++) {
        rowToBeChanged = (startrow + l) % height;
        currentRow = Ab[rowToBeChanged];
        currentRow_changed = currentRow;

        randomNumber = fast_kiss32(state);
        for (int i = 0; i < DIM_PARAM; i++)
            currentRow_changed ^= (randomNumber >> i) & 11 ?(0 << i) : (1 << i);
        
        error = 0;

#pragma omp parallel for private(cEntryOld, cEntryNew, cTruthEntry) reduction(+: error)
        for (int tid = 0; tid < width; tid++) {
            uint32_t currentCol = Bb[tid];
            int intId = (tid * height + rowToBeChanged) / 32;
            int intLane = (tid * height + rowToBeChanged) % 32;
            cTruthEntry = (C0[intId] >> 32 - intLane - 1) & 1;

            cEntryOld = (currentRow         & currentCol) > 0 ? 1 : 0;
            cEntryNew = (currentRow_changed & currentCol) > 0 ? 1 : 0;
            error += ((cEntryNew - cTruthEntry) * (cEntryNew - cTruthEntry)) -
                     ((cEntryOld - cTruthEntry) * (cEntryOld - cTruthEntry));
        }

        if (error < 0) {
            Ab[rowToBeChanged] = currentRow_changed;
            *hDistance = *hDistance + error;
        } 

    }
}

void CPUvectorMatrixMultCompareCol( uint32_t *Ab, uint32_t *Bb, uint32_t *C0, 
                                    int width, int height, int startcol,
                                    int *hDistance, fast_kiss_state32_t *state, int rowsAtOnce) {
    int colToBeChanged = startcol;
    int error;
    int cTruthEntry;
    int cEntryOld;
    int cEntryNew;
    uint32_t currentCol;
    uint32_t currentCol_changed;
    uint32_t randomNumber;

    // Change multiple cols from B
    for (int l = 0; l < rowsAtOnce; l++) {
        colToBeChanged = (colToBeChanged + l) % width;
        currentCol = Bb[colToBeChanged];
        currentCol_changed = currentCol;

        randomNumber = fast_kiss32(state);
        for (int i = 0; i < DIM_PARAM; i++)
            currentCol_changed ^= (randomNumber >> i) & 11 ? (0 << i) : (1 << i);
        
        error = 0;
        #pragma omp parallel for private(cEntryOld, cEntryNew, cTruthEntry) reduction(+: error)
        for (int tid = 0; tid < height; tid++) {
             uint32_t currentRow = Ab[tid]; 
            int intId = (colToBeChanged * height + tid) / 32;
            int intLane = (colToBeChanged * height + tid) % 32;
            cTruthEntry = (C0[intId] >> 32 - intLane - 1) & 1;

            cEntryOld = (currentCol         & currentRow) > 0 ? 1 : 0;
            cEntryNew = (currentCol_changed & currentRow) > 0 ? 1 : 0;
            error += ((cEntryNew - cTruthEntry) * (cEntryNew - cTruthEntry)) - 
                     ((cEntryOld - cTruthEntry) * (cEntryOld - cTruthEntry));
        }

        if (error < 0) {
            Bb[colToBeChanged] = currentCol_changed;
            *hDistance = *hDistance + error;
        }              
    }
}

// Used for debugging and checking, not optimized
void aftertestCPU(  uint32_t *Ab, uint32_t *Bb, uint32_t *d_Ab, uint32_t *d_Bb, uint32_t *C0b, 
                    int width, int height) {    
    TIMERSTART(aftertestCPU)
    int *A, *B, *C0;
    int *d_A, *d_B;
    uint32_t *C_test_CPU;
    uint32_t *d_C_test_CPU;
    A = (int*)malloc(sizeof(int) * DIM_PARAM * height);
    B = (int*)malloc(sizeof(int) * width * DIM_PARAM);
    C0 = (int*)malloc(sizeof(int) * width * height);
    C_test_CPU = (uint32_t *) malloc(sizeof(uint32_t) * width * height);                CUERR
    hipMalloc((void**)&d_A, sizeof(int) * DIM_PARAM * height);                         CUERR
    hipMalloc((void**)&d_B, sizeof(int)*width * DIM_PARAM);                            CUERR
    hipMalloc((void**) &d_C_test_CPU, sizeof(uint32_t) * height * width);              CUERR

    
    for(int i=0; i<height;i++)
        for(int j=0;j<DIM_PARAM;j++)
            A[i*DIM_PARAM + j] = (Ab[i] >> DIM_PARAM-j-1) & 1;

    for(int i=0;i<width;i++)
        for(int j=0;j<DIM_PARAM;j++)
            B[j*width+i] = (Bb[i] >> DIM_PARAM-j-1) & 1;
        
    int intId;
    int intLane;
    for(int i=0; i<width; i++){
        for(int j=0;j<height;j++){
             intId = (i*height + j) / 32;
             intLane = (i*height + j) % 32;
             C0[j*width + i] = (C0b[intId] >> 32 - intLane - 1) & 1;
        }
    }

    
    hipMemcpy(d_A, A, sizeof(uint32_t) * height * DIM_PARAM, hipMemcpyHostToDevice);  CUERR
    hipMemcpy(d_B, B, sizeof(uint32_t) * width * DIM_PARAM, hipMemcpyHostToDevice);   CUERR
    hipMemcpy(d_Ab, Ab, sizeof(uint32_t) * height, hipMemcpyHostToDevice);            CUERR
    hipMemcpy(d_Bb, Bb, sizeof(uint32_t) * width, hipMemcpyHostToDevice);             CUERR
    
    // Doing a check two times: once with A,B and once with Ab,Bb just to make sure
    
    matrixMultiply <<< width / THREADSPERBLOCK + 1, THREADSPERBLOCK >>> 
                        (d_Ab, d_Bb, d_C_test_CPU, width, height);                      CUERR
    
    hipMemcpy(C_test_CPU, d_C_test_CPU, sizeof(uint32_t) * height * width, 
                    hipMemcpyDeviceToHost);                                            CUERR
                    
    int distance_test_CPU = 0;
    for (int i = 0; i < height * width; i++)
        distance_test_CPU += ((C0[i] - C_test_CPU[i]) * (C0[i] - C_test_CPU[i]));
    
    dim3 dimGrid((width-1)/TILE_WIDTH+1, (height-1)/TILE_WIDTH+1, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    matrixMultiplyInt <<< dimGrid, dimBlock >>> 
                (d_A, d_B, d_C_test_CPU, height, width, DIM_PARAM);                     CUERR
    hipMemcpy(C_test_CPU, d_C_test_CPU, sizeof(int) * height * width, 
                    hipMemcpyDeviceToHost);                                            CUERR
    int distance_test_CPU_2 = 0;
    for (int i = 0; i < height * width; i++) 
        distance_test_CPU_2 += ((C0[i] - C_test_CPU[i]) * (C0[i] - C_test_CPU[i]));
    
    TIMERSTOP(aftertestCPU)
    printf("Aftertest error between C0 and C on CPU (bitwise): %i\n", distance_test_CPU);
    printf("Aftertest error between C0 and C on CPU (float): %i\n", distance_test_CPU_2);
}

/*
void initializeTextureMemory(uint32_t **C0, int width, int height) {
    // Texture Memory Initialization
    //////////////////////////////////////////////////////////////////////////////////////
    /*
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    CUERR
    hipArray *cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);
    CUERR
    hipMemcpyToArray(cuArray, 0, 0, *C0, sizeof(float) * width * height, hipMemcpyHostToDevice);
    CUERR
    
    texRef.addressMode[0] = hipAddressModeWrap;
    texRef.addressMode[1] = hipAddressModeWrap;
    hipBindTextureToArray(texRef, cuArray, channelDesc);
    
    uint* d_C0_texture;
    size_t pitch;
    hipMallocPitch((void**)&d_C0_texture, &pitch, width*sizeof(uint32_t), height);
    hipMemcpy2D(d_C0_texture, pitch, *C0, width*sizeof(uint32_t), width*sizeof(uint32_t), height, hipMemcpyHostToDevice);
    
    hipChannelFormatDesc desc = hipCreateChannelDesc<uint>(); 
    hipBindTexture2D(NULL, texRef, d_C0_texture, desc, width, height, pitch);
    ///////////////////////////////////////////////////////////////////////////////////////
}*/


